#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <chrono>
#include <cassert>

#define ROW_NUM 16
#define COL_NUM 256

std::vector<float> init_random_vector(int length) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);
    std::vector<float> random_vector(length);

    for (int i = 0; i < length; i++) {
        random_vector[i] = dis(gen);
    }
    return random_vector;
}

std::vector<float> cpu_verification(const std::vector<float>& in, int row, int col) {
    assert(row * col == in.size());
    std::vector<float> ret;
    for (int i = 0; i < row; i++) {
        float temp = 0.f;
        for (int j = 0; j < col; j ++) {
            temp += in[i * col + j];
        }
        ret.push_back(temp);
    }
    return ret;
}

template <typename T>
__global__ void reduction0(T* __restrict__ in, T* __restrict__ out) {
    extern __shared__ T sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    sdata[tid] = in[i];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}

// split one line one block here into one line two block
template <typename T>
__global__ void reduction1(T* __restrict__ in, T* __shared__ out) {
    extern __shared__ T sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i   = blockIdx.x * blockDim.x * 2 + tid;
    sdata[tid] = in[i] + in[i + blockDim.x];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}



int main() {
    std::vector<float> h_in = init_random_vector(ROW_NUM * COL_NUM);
    std::vector<float> h_out(ROW_NUM, 0.f);

    float *d_in, *d_out;
    hipMalloc(&d_in, sizeof(float) * ROW_NUM * COL_NUM);
    hipMalloc(&d_out, sizeof(float) * ROW_NUM);

    hipMemcpy(d_in, h_in.data(), sizeof(float) * h_in.size(), hipMemcpyHostToDevice);
    // reduction0<<<ROW_NUM, COL_NUM, COL_NUM * sizeof(float)>>>(d_in, d_out);
    reduction1<<<ROW_NUM, COL_NUM / 2, COL_NUM / 2 * sizeof(float)>>>(d_in, d_out);
    auto cpu_result = cpu_verification(h_in, ROW_NUM, COL_NUM);


    hipDeviceSynchronize();
    hipMemcpy(h_out.data(), d_out, sizeof(float) * h_out.size(), hipMemcpyDeviceToHost);

    for (int i = 0; i < h_out.size(); i++) {
        std::cout << "cuda : " <<h_out[i] << ", cpu : " << cpu_result[i] << std::endl;
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
